#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */

#include <stdio.h>

#define BLOCK_SIZE 32
#define GRID_SIZE 200

//extern "C" {
#include "cuda-rsa.h"
//}


/*This macro was taken from the book CUDA by example.*/
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/*This function shifts x to the right by one bit.*/
__device__ void parallelShiftR1(uint32_t *x) {
   unsigned int carry;

   if (threadIdx.x) {
      carry = x[threadIdx.x - 1];
   }
   
   x[threadIdx.x] = (x[threadIdx.x] >> 1) | (carry << 31);
}


/* Returns 1 if x >= y*/
__device__ int parallelGeq(uint32_t *x, uint32_t *y) {
   __shared__ int pos;

   if (threadIdx.x == 0) {
      pos = 31;
   }

   if (x[threadIdx.x] != y[threadIdx.x]) {
      atomicMin(&pos, threadIdx.x);
   }
   return x[pos] >= y[pos];
}

/*This function doese parrallel subtraction on x and y and stores the 
  result into result.*/
__device__  void parallelSubtract(uint32_t *result, uint32_t *x, 
 uint32_t *y) {
   __shared__ uint32_t borrows[32];
   uint32_t t;

   if (!threadIdx.x) 
      borrows[31] = 0;
   
   t = x[threadIdx.x] - y[threadIdx.x];

   if (threadIdx.x)
      borrows[threadIdx.x - 1] = (t > x[threadIdx.x]);

   while (__any(borrows[threadIdx.x])) {
      if (borrows[threadIdx.x]) {
         t--;
      }
      
      if (threadIdx.x) {
         borrows[threadIdx.x - 1] = (t == 0xffffffffU && 
          borrows[threadIdx.x]);
      }
   }
   result[threadIdx.x] = t;
}

__device__ int gcd(uint32_t *x, uint32_t *y) {
   
   while (__any(x[threadIdx.x])) {
      while ((x[31] & 1) == 0) {
         parallelShiftR1(x);
      }
      while ((y[31] & 1) == 0) {
         parallelShiftR1(y);
      }
      if (parallelGeq(x, y)) {
         parallelSubtract(x, x, y);
         parallelShiftR1(x);
      }
      else {
         parallelSubtract(y, y, x);
         parallelShiftR1(y);
      }  
   }
   parallelShiftR1(y);
   return __any(y[threadIdx.x]);
}

__global__ void doGCD(bigInt *keys, int toComp, int start, 
 uint32_t *vector) {
   bigInt x, y;
   
   if (start + blockIdx.x < NUM_KEYS) {
      x = keys[toComp];
      y = keys[start + blockIdx.x];
      if (gcd(x.values, y.values)) {
         atomicOr(vector + ((blockIdx.x + start) / 32), 1 << 
          (blockIdx.x % 32));
      }
   }
}

/*Sets up the GPU for the kernel call.*/
extern "C"
void setUpKernel(bigInt *arr, uint32_t *bitVector) {
   dim3 dimGrid(GRID_SIZE);
   dim3 dimBlock(BLOCK_SIZE);

   bigInt *arrD;
   uint32_t *bitVectorD;
   
   int count = 1, ndx = 0;
   int keyArrSize = sizeof(bigInt) * NUM_KEYS; 
   int bitVecSize = sizeof(uint32_t) * INT_ARRAY_SIZE;  

   /*Allocate space on device for bitMatrix, and keys*/
   HANDLE_ERROR(hipMalloc(&arrD, keyArrSize));
   HANDLE_ERROR(hipMalloc(&bitVectorD, bitVecSize));

   /*Copy keys onto device*/
   HANDLE_ERROR(hipMemcpy(arrD, arr, keyArrSize, hipMemcpyHostToDevice));

   while(ndx < NUM_KEYS - 1) {
      doGCD<<<dimGrid, dimBlock>>>(arrD, ndx, count, bitVectorD);
      if (count += GRID_SIZE > NUM_KEYS) {
         ndx++;
         count = ndx + 1;
      }
   }

   HANDLE_ERROR(hipMemcpy(bitVector, bitVectorD, bitVecSize, 
    hipMemcpyDeviceToHost)); 
}

