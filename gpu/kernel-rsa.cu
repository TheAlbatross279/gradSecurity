#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */

#include <stdio.h>
#include "./cuda-rsa-master/src/mpz/mpz.h"

#define ROW_SIZE 8
#define KEYS_PER_KERNEL 64

extern "C" {
#include "cuda-rsa.h"
}


/*This macro was taken from the book CUDA by example.*/
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void findGCD(mpz_t *arrD, int keyPos, int *bitRowD) {
   /*Move key to be gcd'd into sharedMem*/
   __shared__ mpz_t key;
   mpz_init(&key);
   mpz_set(&key, &arrD[keyPos]);
   int toComp = blockIdx.x * blockDim.x + threadIdx.x;
   int mask = 1 << (toComp % sizeof(int));
   mpz_t a, b, c, quo, one;
   
   mpz_init(&one);
   mpz_init(&a);
   mpz_init(&b);
   mpz_set_ui(&one, 1);
   mpz_init(&quo);
   mpz_init(&c);
   mpz_set(&a, &key);
   mpz_set(&b, &arrD[toComp + keyPos]);

   if (toComp + keyPos > keyPos && toComp + keyPos < NUM_KEYS) {
      while(!digits_is_zero(a.digits, a.capacity)) {
         mpz_set(&c, &a);
         mpz_div(&quo, &a, &b, &a);
         mpz_set(&b, &c);
      }
     
      if (mpz_compare(&one, &b)) {
      /*GCD greater than one was found*/
         atomicOr(&bitRowD[toComp / sizeof(int)], mask);
      }
   }
   
}

/*Sets up the GPU for the kernel call.*/
extern "C"
void setUpKernel(mpz_t *arr, int *bitMatrix) {
   /*Set up kernel to run 200000 threads*/
   dim3 dimGrid(8);
   dim3 dimBlock(8);

   mpz_t *arrD;
   int *bitRowD;
   int count = 0, keyArrSize = sizeof(mpz_t) * NUM_KEYS;
   //int rowSize = 80; //sizeof(char) * BYTE_ARRAY_SIZE;
   int byteOffset = 0;
   
   /*Allocate space on device for bitMatrix, and keys*/
   HANDLE_ERROR(hipMalloc(&arrD, keyArrSize));
   HANDLE_ERROR(hipMalloc(&bitRowD, ROW_SIZE));
   
   /*Copy keys onto device*/
   HANDLE_ERROR(hipMemcpy(arrD, arr, keyArrSize, hipMemcpyHostToDevice));
     
   while(count < NUM_KEYS) {
      /*Clear the bit vector row*/
      HANDLE_ERROR(hipMemset(bitRowD, 0, ROW_SIZE));
      
      /*Launch Kernel*/
      printf("launching kernel\n");
      findGCD<<<dimGrid, dimBlock>>>(arrD, count, bitRowD);
      printf("back from kernel\n");
      /*Copy computed bit vector into bit matrix*/
      HANDLE_ERROR(hipMemcpy(bitMatrix + byteOffset, bitRowD, ROW_SIZE, 
       hipMemcpyDeviceToHost));
      byteOffset += 2;
      count += KEYS_PER_KERNEL;
   }
}

