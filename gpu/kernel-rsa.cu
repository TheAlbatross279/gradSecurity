#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */

#include <stdio.h>

#define ROW_SIZE 8
#define KEYS_PER_KERNEL 64

extern "C" {
#include "cuda-rsa.h"
}


/*This macro was taken from the book CUDA by example.*/
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/*This function shifts x to the right by one bit.*/
__device__ void parallelShiftR1(uint32_t *x) {
   unsigned int carry;

   if (threadIdx.x) {
      carry = x[threadIdx.x - 1];
   }
   
   x[threadIdx.x] = (x[threadIdx.x] >> 1) | (carry << 31);
}


/* Returns 1 if x >= y*/
__device__ int parallelGeq(uint32_t *x, uint32_t *y) {
   int pos;

   if (threadIdx.x == 0) {
      pos = 31;
   }

   if (x[threadIdx.x] != y[threadIdx.x]) {
      atomicMin(&pos, threadIdx.x);
   }
   return x[pos] >= y[pos];
}

/*This function doese parrallel subtraction on x and y and stores the 
  result into result.*/
__device__ void parallelSubtract(uint32_t *result, uint32_t *x, 
 uint32_t *y) {
   uint32_t borrows[32];
   uint32_t t;

   if (!threadIdx.x) 
      borrows[31] = 0;
   
   t = x[threadIdx.x] - y[threadIdx.x];

   if (threadIdx.x)
      borrows[threadIdx.x - 1] = (t > x[threadIdx.x]);

   while (__any(borrows[threadIdx.x])) {
      if (borrows[threadIdx.x]) {
         t--;
      }
      
      if (threadIdx.x) {
         borrows[threadIdx.x - 1] = (t == 0xffffffffU && 
          borrows[threadIdx.x]);
      }
   }
   result[threadIdx.x] = t;
}

__global__ int gcd(uint32_t *x, uint32_t *y) {
   
   while (__any(x[threadIdx.x])) {
      while ((x[31] & 1) == 0) {
         parallelShiftR1(x);
      }
      while ((y[31] & 1) == 0) {
         parallelShiftR1(y);
      }
      if (parallelGeq(x, y)) {
         parallelSubtract(x, x, y);
         parallelShiftR1(x);
      }
      else {
         parallelSubtract(y, y, x);
         parallelShiftR1(y);
      }
      
   }
   
}
/*Sets up the GPU for the kernel call.*/
extern "C"
void setUpKernel(bigInt *arr, uint32_t *bitVector) {
   /*Set up kernel to run 200000 threads*/
   dim3 dimGrid(8);
   dim3 dimBlock(8);

   bigInt *arrD;
   uint32_t *bitRowD;

   int count = 0;
   int byteOffset = 0;
   int ndx = 0;
   
   /*Allocate space on device for bitMatrix, and keys*/
   HANDLE_ERROR(hipMalloc(&arrD, keyArrSize));
   HANDLE_ERROR(hipMalloc(&bitRowD, ROW_SIZE));
   
   /*Copy keys onto device*/
   HANDLE_ERROR(hipMemcpy(arrD, arr, keyArrSize, hipMemcpyHostToDevice));
     
   while(ndx < NUM_KEYS) {
      /*Clear the bit vector row*/
      HANDLE_ERROR(hipMemset(bitRowD, 0, ROW_SIZE));
      
      /*Launch Kernel*/
      printf("launching kernel\n");
      findGCD<<<dimGrid, dimBlock>>>(arrD, ndx, bitRowD);
      printf("back from kernel\n");
      /*Copy computed bit vector into bit matrix*/
      HANDLE_ERROR(hipMemcpy(bitMatrix + byteOffset, bitRowD, ROW_SIZE, 
       hipMemcpyDeviceToHost));
      byteOffset += 2;
      count += KEYS_PER_KERNEL;
      if (count > NUM_KEYS) {
         count = 0; 
         ndx++;
      }
   }
}

