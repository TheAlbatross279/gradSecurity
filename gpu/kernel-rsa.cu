#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */

#include <stdio.h>

#define BLOCK_SIZE 32
#define GRID_SIZE 200

//extern "C" {
#include "cuda-rsa.h"
//}


/*This macro was taken from the book CUDA by example.*/
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/*This function shifts x to the right by one bit.*/
__global__ void parallelShiftR1(uint32_t *x) {
   unsigned int carry;

   if (threadIdx.x) {
      carry = x[threadIdx.x - 1];
   }
   
   x[threadIdx.x] = (x[threadIdx.x] >> 1) | (carry << 31);
}


/* Returns 1 if x >= y*/
__device__ int parallelGeq(uint32_t *x, uint32_t *y) {
   __shared__ int pos;

   if (threadIdx.x == 0) {
      pos = 31;
   }

   if (x[threadIdx.x] != y[threadIdx.x]) {
      atomicMin(&pos, threadIdx.x);
   }
   return x[pos] >= y[pos];
}

/*This function doese parrallel subtraction on x and y and stores the 
  result into result.*/
__device__  void parallelSubtract(uint32_t *result, uint32_t *x, 
 uint32_t *y) {
   __shared__ uint32_t borrows[32];
   uint32_t t;

   if (!threadIdx.x) 
      borrows[31] = 0;
   
   t = x[threadIdx.x] - y[threadIdx.x];

   if (threadIdx.x)
      borrows[threadIdx.x - 1] = (t > x[threadIdx.x]);

   while (__any(borrows[threadIdx.x])) {
      if (borrows[threadIdx.x]) {
         t--;
      }
      
      if (threadIdx.x) {
         borrows[threadIdx.x - 1] = (t == 0xffffffffU && 
          borrows[threadIdx.x]);
      }
   }
   result[threadIdx.x] = t;
}

__device__ int gcd(uint32_t *x, uint32_t *y) {
   
   /*while (__any(x[threadIdx.x])) {
      while ((x[31] & 1) == 0) {
         parallelShiftR1(x);
      }
      while ((y[31] & 1) == 0) {
         parallelShiftR1(y);
      }
      if (parallelGeq(x, y)) {
         parallelSubtract(x, x, y);
         parallelShiftR1(x);
      }
      else {
         parallelSubtract(y, y, x);
         parallelShiftR1(y);
      }  
   }
   parallelShiftR1(y);*/
   return __any(y[threadIdx.x]);
}


bigInt testKernel(bigInt x) {
   dim3 dimGrid(1);
   dim3 dimBlock(32);

   bigInt *xD;
   bigInt result;
   int *rD;

   HANDLE_ERROR(hipMalloc(&xD, sizeof(bigInt)));     

   HANDLE_ERROR(hipMemcpy(xD, &x, sizeof(bigInt), hipMemcpyHostToDevice));
   
   parallelShiftR1<<<dimGrid, dimBlock>>>(xD->values);

   HANDLE_ERROR(hipMemcpy(&result, xD, sizeof(bigInt), 
    hipMemcpyDeviceToHost));

   hipFree(xD);

   return result;
}
/*Sets up the GPU for the kernel call.*/
extern "C"
void setUpKernel(bigInt *arr, uint32_t *bitVector) {
   dim3 dimGrid(GRID_SIZE);
   dim3 dimBlock(BLOCK_SIZE);

   bigInt *arrD;
   uint32_t *bitRowD;
   
   int count = 0, ndx = 0, byteOffset = 0;
   int keyArrSize = sizeof(bigInt) * NUM_KEYS;   

   /*Allocate space on device for bitMatrix, and keys*/
   HANDLE_ERROR(hipMalloc(&arrD, sizeof(bigInt) * NUM_KEYS));
   HANDLE_ERROR(hipMalloc(&bitRowD, sizeof(uint32_t) * INT_ARRAY_SIZE));

   /*Copy keys onto device*/
   HANDLE_ERROR(hipMemcpy(arrD, arr, keyArrSize, hipMemcpyHostToDevice));

   while(ndx < NUM_KEYS) {
      /*Clear the bit vector row*/
      //HANDLE_ERROR(hipMemset(bitRowD, 0, ROW_SIZE));

      /*Launch Kernel*/
      printf("launching kernel\n");
      //findGCD<<<dimGrid, dimBlock>>>(arrD, ndx, bitRowD);
      printf("back from kernel\n");
      /*Copy computed bit vector into bit matrix*/
      //HANDLE_ERROR(hipMemcpy(bitMatrix + byteOffset, bitRowD, ROW_SIZE, 
       //hipMemcpyDeviceToHost));
      byteOffset += 2;
      //count += KEYS_PER_KERNEL;
      if (count > NUM_KEYS) {
         count = 0; 
         ndx++;
      }
   }
}

