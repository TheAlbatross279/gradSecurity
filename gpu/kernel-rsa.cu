#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */

#include <stdio.h>
#include "./cuda-rsa-master/src/mpz/mpz.h"

#define ROW_SIZE 8
#define KEYS_PER_KERNEL 64

extern "C" {
#include "cuda-rsa.h"
}


/*This macro was taken from the book CUDA by example.*/
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/*Device function that returns zero if the int passed is zero and 1 if
  the int passed is nonzero*/
__device__ int parallelNonZero(int *x) {
   if (__any(x[threadIdx.x]))
      return 1;
   return 0;
}


/*This function doese parrallel subtraction on x and y and stores the 
  result into result.*/
__device__ void parallelSubtract(int *result, int *x, int *y) {
   unsigned int borrows[32];
   unsigned int t;

   if (!threadIdx.x) 
      borrows[31] = 0;
   
   t = x[threadIdx.x] - y[threadIdx.x];

   if (threadIdx.x)
      borrows[threadIdx.x - 1] = (t > x[threadIdx.x]);

   while (parallelNonZero(borrows)) {
      if (borrows[threadIdx.x]) {
         t--;
      }
      
      if (threadIdx.x) {
         borrows[threadIdx.x - 1] = (t == 0xffffffffU && 
          borrows[threadIdx.x]);
      }
   }
   result[threadIdx.x] = t;
}

__global__ void findGCD(mpz_t *arrD, int keyPos, int *bitRowD) {
   /*Move key to be gcd'd into sharedMem*/
   __shared__ mpz_t key;
   mpz_init(&key);
   mpz_set(&key, &arrD[keyPos]);
   int toComp = blockIdx.x * blockDim.x + threadIdx.x;
   int mask = 1 << (toComp % sizeof(int));
   mpz_t a, b, c, quo, one;
   
   mpz_init(&one);
   mpz_init(&a);
   mpz_init(&b);
   mpz_set_ui(&one, 1);
   mpz_init(&quo);
   mpz_init(&c);
   mpz_set(&a, &key);
   mpz_set(&b, &arrD[toComp + keyPos]);

   if (toComp + keyPos > keyPos && toComp + keyPos < NUM_KEYS) {
      while(!digits_is_zero(a.digits, a.capacity)) {
         mpz_set(&c, &a);
         mpz_div(&quo, &a, &b, &a);
         mpz_set(&b, &c);
      }
     
      if (mpz_compare(&one, &b)) {
      /*GCD greater than one was found*/
         atomicOr(&bitRowD[toComp / sizeof(int)], mask);
      }
   }
   
}

/*Sets up the GPU for the kernel call.*/
extern "C"
void setUpKernel(mpz_t *arr, int *bitMatrix) {
   /*Set up kernel to run 200000 threads*/
   dim3 dimGrid(8);
   dim3 dimBlock(8);

   mpz_t *arrD;
   int *bitRowD;
   int count = 0, keyArrSize = sizeof(mpz_t) * NUM_KEYS;
   //int rowSize = 80; //sizeof(char) * BYTE_ARRAY_SIZE;
   int byteOffset = 0;
   int ndx = 0;
   
   /*Allocate space on device for bitMatrix, and keys*/
   HANDLE_ERROR(hipMalloc(&arrD, keyArrSize));
   HANDLE_ERROR(hipMalloc(&bitRowD, ROW_SIZE));
   
   /*Copy keys onto device*/
   HANDLE_ERROR(hipMemcpy(arrD, arr, keyArrSize, hipMemcpyHostToDevice));
     
   while(ndx < NUM_KEYS) {
      /*Clear the bit vector row*/
      HANDLE_ERROR(hipMemset(bitRowD, 0, ROW_SIZE));
      
      /*Launch Kernel*/
      printf("launching kernel\n");
      findGCD<<<dimGrid, dimBlock>>>(arrD, ndx, bitRowD);
      printf("back from kernel\n");
      /*Copy computed bit vector into bit matrix*/
      HANDLE_ERROR(hipMemcpy(bitMatrix + byteOffset, bitRowD, ROW_SIZE, 
       hipMemcpyDeviceToHost));
      byteOffset += 2;
      count += KEYS_PER_KERNEL;
      if (count > NUM_KEYS) {
         count = 0; 
         ndx++;
      }
   }
}

