#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */

#include <stdio.h>
#include "./cuda-rsa-master/src/mpz/mpz.h"

extern "C" {
#include "cuda-rsa.h"
}


/*This macro was taken from the book CUDA by example.*/
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void findGCD(mpz_t *arrD, int keyPos, char *bitRowD) {
   /*Move key to be gcd'd into sharedMem*/
   __shared__ mpz_t key;
   mpz_init(&key);
   mpz_set(&key, &arrD[keyPos]);
   int toComp = blockIdx.x * blockDim.x + threadIdx.x;
   char mask = 1 << (toComp % sizeof(char));
   mpz_t a, b, c, quo, one;
   
   mpz_init(&one);
   mpz_init(&a);
   mpz_init(&b);
   mpz_set_ui(&one, 1);
   mpz_init(&quo);
   mpz_init(&c);
   mpz_set(&a, &key);
   mpz_set(&b, &arrD[toComp]);

   if (toComp > keyPos) {
      while(!digits_is_zero(a.digits, a.capacity)) {
         mpz_set(&c, &a);
         mpz_div(&quo, &a, &b, &a);
         mpz_set(&b, &c);
      }
     
      if (mpz_compare(&one, &b)) {
      /*GCD greater than one was found*/
         bitRowD[toComp / sizeof(char)] |= mask;
      }
   }
   
}

/*Sets up the GPU for the kernel call.*/
extern "C"
void setUpKernel(mpz_t *arr, char *bitMatrix) {
   /*Set up kernel to run 200000 threads*/
   dim3 dimGrid(2000);
   dim3 dimBlock(100);

   mpz_t *arrD;
   char *bitRowD;
   int count = 0, keyArrSize = sizeof(mpz_t) * NUM_KEYS;
   int rowSize = sizeof(char) * BYTE_ARRAY_SIZE;
   
   /*Allocate space on device for bitMatrix, and keys*/
   HANDLE_ERROR(hipMalloc(&arrD, keyArrSize));
   HANDLE_ERROR(hipMalloc(&bitRowD, rowSize));
   
   /*Copy keys onto device*/
   HANDLE_ERROR(hipMemcpy(arrD, arr, keyArrSize, hipMemcpyHostToDevice));
     
   while(count < NUM_KEYS) {
      /*Clear the bit vector row*/
      HANDLE_ERROR(hipMemset(bitRowD, 0, rowSize));
      
      /*Launch Kernel*/
      printf("launching kernel\n");
      findGCD<<<dimGrid, dimBlock>>>(arrD, count, bitRowD);
      printf("back from kernel\n");
      /*Copy computed bit vector into bit matrix*/
      HANDLE_ERROR(hipMemcpy(bitMatrix + (count * BYTE_ARRAY_SIZE), 
       bitRowD, rowSize, hipMemcpyDeviceToHost));

      count = NUM_KEYS;
   }
}

