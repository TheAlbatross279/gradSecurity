#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */

#include <stdio.h>

#define BLOCK_SIZE 32
#define GRID_SIZE 4096

#include "cuda-rsa.h"


/*This macro was taken from the book CUDA by example.*/
static void HandleError(hipError_t err, const char *file, int line ) { 
   if (err != hipSuccess) {
      printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
      exit(1);
   }   
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


/*This function shifts x to the right by one bit.*/
__device__ void parallelShiftR1(uint32_t *x) {
   unsigned int carry;

   if (threadIdx.x) {
      carry = x[threadIdx.x - 1];
   }
   
   x[threadIdx.x] = (x[threadIdx.x] >> 1) | (carry << 31);
}


/* Returns 1 if x >= y*/
__device__ int parallelGeq(uint32_t *x, uint32_t *y) {
   __shared__ int pos;

   if (threadIdx.x == 0) {
      pos = 31;
   }

   if (x[threadIdx.x] != y[threadIdx.x]) {
      atomicMin(&pos, threadIdx.x);
   }
   return x[pos] >= y[pos];
}

/*This function doese parrallel subtraction on x and y and stores the 
  result into result.*/
__device__  void parallelSubtract(uint32_t *result, uint32_t *x, 
 uint32_t *y) {
   __shared__ uint32_t borrows[32];
   uint32_t t;

   if (!threadIdx.x) {
      borrows[31] = 0;
   }
   
   t = x[threadIdx.x] - y[threadIdx.x];

   if (threadIdx.x) {
      borrows[threadIdx.x - 1] = (t > x[threadIdx.x]);
   }

   while (__any(borrows[threadIdx.x])) {
      if (borrows[threadIdx.x]) {
         t--;
      }
      
      if (threadIdx.x) {
         borrows[threadIdx.x - 1] = (t == 0xffffffffU && 
          borrows[threadIdx.x]);
      }
   }
   result[threadIdx.x] = t;
}

__device__ void gcd(uint32_t *x, uint32_t *y, int *res) {
   /*__shared__ uint32_t x[32];
   __shared__ uint32_t y[32];

   x[threadIdx.x] = x1[threadIdx.x];
   y[threadIdx.x] = y1[threadIdx.y];*/
   
   while (__any(x[threadIdx.x])) {
      while ((x[31] & 1) == 0) {
         parallelShiftR1(x);
      }
      while ((y[31] & 1) == 0) {
         parallelShiftR1(y);
      }
      if (parallelGeq(x, y)) {
         parallelSubtract(x, x, y);
         parallelShiftR1(x);
      }
      else {
         parallelSubtract(y, y, x);
         parallelShiftR1(y);
      }  
   }
   parallelShiftR1(y);
   *res = __any(y[threadIdx.x]);
}

__global__ void doGCD(bigInt *keys, int toComp, int start, 
 uint32_t *vector, bigInt *comp, bigInt *comp2) {
   int res;
   if (start + blockIdx.x < NUM_KEYS) {
      comp[blockIdx.x].values[threadIdx.x] = 
       keys[toComp].values[threadIdx.x];
      comp2[blockIdx.x].values[threadIdx.x] = 
       keys[start + blockIdx.x].values[threadIdx.x];
      gcd(comp[blockIdx.x].values, comp2[blockIdx.x].values, &res);
      if (res) {
         if (threadIdx.x == 0) {
           //printf("FOUND A KEY at: %d, %d\n", toComp, start + blockIdx.x);
            atomicOr(&vector[(blockIdx.x + start) / 32], 1 << 
             ((blockIdx.x + start) % 32));
            atomicOr(&vector[toComp / 32], 1 << (toComp % 32));
         }
      }
   }
}

/*Sets up the GPU for the kernel call.*/
int setUpKernel(bigInt *arr, uint32_t *bitVector, int **indexs) {
   dim3 dimGrid(GRID_SIZE);
   dim3 dimBlock(BLOCK_SIZE);

   bigInt *arrD, *compD, *comp2D;
   uint32_t *bitVectorD;

   
   int count = 1, ndx = 0;
   int keyArrSize = sizeof(bigInt) * NUM_KEYS; 
   int bitVecSize = sizeof(uint32_t) * INT_ARRAY_SIZE;  
   
   /*Allocate space on device for bitMatrix, and keys*/
   HANDLE_ERROR(hipMalloc(&arrD, keyArrSize));
   HANDLE_ERROR(hipMalloc(&bitVectorD, bitVecSize)); 
   HANDLE_ERROR(hipMalloc(&compD, sizeof(bigInt) * GRID_SIZE));
   HANDLE_ERROR(hipMalloc(&comp2D, sizeof(bigInt) * GRID_SIZE));
   HANDLE_ERROR(hipMemset(bitVectorD, 0, bitVecSize));
   
   /*Copy keys onto device*/
   HANDLE_ERROR(hipMemcpy(arrD, arr, keyArrSize, hipMemcpyHostToDevice));

   while(ndx < NUM_KEYS - 1) {
      //printf("ndx: %d, start: %d\n", ndx, count);
      doGCD<<<dimGrid, dimBlock>>>(arrD, ndx, count, bitVectorD, compD, comp2D);
      count += GRID_SIZE;
      if (count >= NUM_KEYS) {
         ndx++;
         count = ndx + 1;
      }
   }

   HANDLE_ERROR(hipMemcpy(bitVector, bitVectorD, bitVecSize, 
    hipMemcpyDeviceToHost));
   
   uint32_t mask;
   int total = 0;
   int inCount = 0;

   for (count = 0; count < INT_ARRAY_SIZE; count++) {
      for (inCount = 0; inCount < 32; inCount++) {
         mask = 1 << inCount;
         if (bitVector[count] & mask) {
            total++;
            //printf("key location: %d\n", (count * 32) + inCount);
            //printf("count: %d, inCound: %d\n", count, inCount);
            
         }
      }
   }
   *indexs = (int *)calloc(total, sizeof(int));
   if (!indexs) {
      perror("calloc");
      exit(1);
   }

   int pairs = 0;
   for (count = 0; count < INT_ARRAY_SIZE; count++) {
      for (inCount = 0; inCount < 32; inCount++) {
         mask = 1 << inCount;
         if (bitVector[count] & mask) {
            //printf("key location: %d\n", (count * 32) + inCount);
            indexs[0][pairs++] = (count * 32) + inCount;
         }
      }
   }

   for (count = 0; count < total; count++) {
      printf("count: %d ndx: %d\n", count, indexs[0][count]);
   }
 
   printf("total: %d\n", total);
   return total;
}

