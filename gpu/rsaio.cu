#include "hip/hip_runtime.h"
/* Brady Thomas & Kim Paterson
 * CSC 419/556 Cuda-RSA 
 * Feb, 2014
 * 
 * Cuda-rsa code licensed from https://github.com/dmatlack/cuda-rsa
 */
//#include "/home/clupo/gmp/mpz.h"
#include <gmp.h>
#include <stdint.h>
#include "rsaio.h"
#include "cuda-rsa.h"

void outputKeys(int *bad_keys, FILE *outfile, mpz_t *arr, int num_bad_keys) {
  int i = 0, j = 0;
  mpz_t gcd;
  mpz_init(gcd);

  mpz_t privateKey;
  mpz_init(privateKey);

  for (i=0; i < num_bad_keys; i++) {
     for (j=i+1; j < num_bad_keys; j++) {
        mpz_gcd (gcd, arr[bad_keys[j]], arr[bad_keys[i]]);
                   
        //if it's not 1, then output
        if (mpz_cmp_ui(gcd, 1) != 0) {
//           printf("%d and %d are bad keys\n", bad_keys[i], bad_keys[j]);
           //generate keys
           generateKeys(gcd, arr[bad_keys[i]], privateKey);
                      
           //output i key and private key
           outputPrivateKey(arr[bad_keys[i]], outfile);
           fprintf(outfile, ":");
           outputPrivateKey(privateKey, outfile);
           fprintf(outfile, "\n");

           //get j private key
           generateKeys(gcd, arr[bad_keys[j]], privateKey);
                
           //output j key and private key
           outputPrivateKey(arr[bad_keys[j]], outfile);
           fprintf(outfile, ":");
           outputPrivateKey(privateKey, outfile);
           fprintf(outfile, "\n");
        }
     }
  
  }
}
    

void outputPrivateKey(mpz_t privateKey, FILE *file) {
  mpz_out_str(file, 10, privateKey);
}

void generateKeys(mpz_t gcd, mpz_t modulii, mpz_t privateKey) {
  //set public key
  mpz_t publicKey;
  mpz_init(publicKey);
  mpz_set_ui(publicKey, E);  
  getPrivateKey(gcd, modulii, publicKey, privateKey);
  
}

void getPrivateKey(mpz_t p, mpz_t modulii, mpz_t publicKey, mpz_t privateKey) {
  mpz_t q;
  mpz_init(q);
  
  mpz_cdiv_q(q, modulii, p);
  //q is now other prime

  //get keys
  getKeysWithPrimes(p, q, publicKey, privateKey);

}


void getKeysWithPrimes(mpz_t p, mpz_t q, mpz_t publicKey, mpz_t privateKey) {
  mpz_t n;
  mpz_init(n);
  //compute totient
  totient(p, q, n);
  
  //compute d/private key
  mpz_t neg_one;
  mpz_init(neg_one);
  mpz_set_si(neg_one, -1);

  //set privateKey
  modExponentMPZ(publicKey, neg_one, n, privateKey);
}

void totient(mpz_t prime1, mpz_t prime2, mpz_t n) {
  //totient(n) = totient(p)*totient(q) = (p-1)(q-1)
  unsigned long int one = 1;
  mpz_t p, q;
  mpz_init(p);
  mpz_init(q);

  //subtract one from each 
  mpz_sub_ui(p, prime1, one);
  mpz_sub_ui(q, prime2, one);

  //(p-1)(q-1)
  mpz_mul(n, p, q);
  
}

void modExponentMPZ(mpz_t base, mpz_t exp, mpz_t mod, mpz_t result) {
  mpz_powm (result, base, exp, mod);
}
